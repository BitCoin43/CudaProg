//#include "DeviceKernel.cuh"
#include <cmath>
#include <algorithm>
#include <thread>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/hip_cooperative_groups.h>

